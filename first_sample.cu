#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//Windows対応はしない

#include <helper_gl.h>
#include <GL/freeglut.h>

#include <helper_functions.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define REFRESH_DELAY     (50) //ms
#define D_ANGLE (5) //ms

#define NUM_OF_NEURONS (96749)
#define PRE_LOAD (50)
#define TAU (4.0)
#define DT (1.0)
/////////////////////////////////////
//constants
const unsigned int window_width = 1024;
const unsigned int window_height = 1024;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

////! VBO variables
GLuint *vbo;
struct hipGraphicsResource** vbo_resources;


/////////////////////////////////////
bool initGL(int *argc, char **argv);

void display();
void display2();

void timerEvent(int value);
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

void createVBOs(GLuint **vbo_list, struct hipGraphicsResource ***vbo_resources, int vbo_num, unsigned int size, unsigned int vbo_res_flags);
void deleteVBOs(GLuint *vbo, struct hipGraphicsResource **vbo_res, int vbo_num);

/////////////////////////////////////

void LoadPos( float4 *pos, char* type_arr, GLfloat frame_vertex[8][3], char *filename, int num );



///////////////////////////////////// CUDA
__global__ void simple_vbo_kernel(float4 *pos, float4 *col, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;
    float c = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f + 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(u, w, v, 1.0f);
    col[y*width+x] = make_float4(c, c, c, 1.0f);
}

__global__ void init_act_spike( float *d_activity, char *d_spike, int num ){
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < num){
        d_activity[id] = 0.f;
        d_spike[id] = 0;
    }
};

__global__ void update_activity(float4 *color, char *type, int3 *c_map, float *activity, char *spike, unsigned int num, int sub_t){
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    float da = 0;
    if(id < num){
        float a = activity[id];
        da = -a/TAU;
        a = ( spike[ num*sub_t + id] )? 1.0: a + da;
        activity[id] = a;
        color[ id ] = make_float4( ((float) c_map[type[id]].x)/255.f*a , ((float) c_map[type[id]].y)/255.f*a, ((float) c_map[type[id]].z)/255.f*a, (a > 0.2)?a:0 );
    }
}
/////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){

#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif

    if(false == initGL(&argc, argv)){
        fprintf(stderr, "ERROR: initGL Error");
        exit(1);
    }


    createVBOs(&vbo, &vbo_resources, 2, NUM_OF_NEURONS * 4 * sizeof(float), cudaGraphicsMapFlagsWriteDiscard);


    glutDisplayFunc(display2);

    glutMainLoop();

    return 0;
}

/////////////////////////////////////////////////////////////////////////

void LoadPos( float4 *pos, char *type_arr, GLfloat frame_vertex[8][3], const char *filename, int num ){
    FILE *fp;
    if( (fp = fopen( filename, "r+")) == NULL ){
        fprintf(stderr, "%s can't open\n", filename);
        exit(1);
    }
    char s[256];
    int c = 0;
    float x_range[2] = {0,0};
    float y_range[2] = {0,0};
    float z_range[2] = {0,0};
    while( fgets(s, 256, fp) != NULL){
        int id, type;
        double x, y, z;
        sscanf(s, "%d %d %lf %lf %lf", &id, &type, &x, &y, &z);
        
        x_range[0] = (x_range[0] > x/100)? x/100: x_range[0];
        x_range[1] = (x_range[1] < x/100)? x/100: x_range[1];
        y_range[0] = (y_range[0] > y/100)? y/100: y_range[0];
        y_range[1] = (y_range[1] < y/100)? y/100: y_range[1];
        z_range[0] = (z_range[0] > z/100)? z/100: z_range[0];
        z_range[1] = (z_range[1] < z/100)? z/100: z_range[1];


        if( id <= num ){
            pos[id-1] = make_float4( x/100, y/100, z/100, 1.0f);
            type_arr[id-1] = type;
        }else{
            fprintf(stderr, "id %d is out of range %d\n", id, num);
        }
        c++;
    }
    fprintf(stderr, "xrange %f - %f\n", x_range[0], x_range[1]);
    fprintf(stderr, "yrange %f - %f\n", y_range[0], y_range[1]);
    fprintf(stderr, "zrange %f - %f\n", z_range[0], z_range[1]);
    

        frame_vertex[ 0 ][0] = x_range[0];
        frame_vertex[ 0 ][1] = y_range[0];
        frame_vertex[ 0 ][2] = z_range[0];

        frame_vertex[ 1 ][0] = x_range[0]; 
        frame_vertex[ 1 ][1] = y_range[0]; 
        frame_vertex[ 1 ][2] = z_range[1]; 

        frame_vertex[ 2 ][0] = x_range[0]; 
        frame_vertex[ 2 ][1] = y_range[1]; 
        frame_vertex[ 2 ][2] = z_range[1]; 

        frame_vertex[ 3 ][0] = x_range[0];
        frame_vertex[ 3 ][1] = y_range[1];
        frame_vertex[ 3 ][2] = z_range[0];

        frame_vertex[ 4+0 ][0] = x_range[1];
        frame_vertex[ 4+0 ][1] = y_range[0];
        frame_vertex[ 4+0 ][2] = z_range[0];

        frame_vertex[ 4+1 ][0] = x_range[1]; 
        frame_vertex[ 4+1 ][1] = y_range[0]; 
        frame_vertex[ 4+1 ][2] = z_range[1]; 

        frame_vertex[ 4+2 ][0] = x_range[1]; 
        frame_vertex[ 4+2 ][1] = y_range[1]; 
        frame_vertex[ 4+2 ][2] = z_range[1]; 

        frame_vertex[ 4+3 ][0] = x_range[1];
        frame_vertex[ 4+3 ][1] = y_range[1];
        frame_vertex[ 4+3 ][2] = z_range[0];

    fprintf(stderr, "%d positions have loaded.\n", c);
    return;
}

void LoadSpike( FILE **fp, char *spike, int num, int width ){
    static int isFirst = true;
    static float old_t = 0.f;
    static int sub_id = 0;
    int count = 0;
    memset( (void*)spike, 0, sizeof(char)*num*width );

    if( !isFirst ){
        spike[sub_id] = 1;
    }

    char s[256];
    char *err_s;
    while( (err_s = fgets(s, 256, *fp) ) != NULL){
        int id;
        double t;

        sscanf(s, "%lf %d %*d", &t, &id);
        if(id >= num){
            fprintf(stderr, "id %d is out of range %d\n ->:%s\n", id, num, err_s);
            continue;
        }
        if( old_t < t ){
            count++;
            old_t = t;
            if( !(count < width) ){
                sub_id = id;
                break;
            }else{
                spike[ num*( ((int)t)%width ) + id] = 1;
            }
        }else{
            spike[ num*( ((int)t)%width ) + id] = 1;
        }
    }

    if(err_s == NULL){
        fclose(*fp);
        *fp = fopen( "spike_data/spike.dat", "r" );
        fprintf(stderr, "file_reset\n");
        isFirst = true;

    }else{
        isFirst = false;
    }

}


/////////////////////////////////////////////////////////////////////////


bool initGL(int *argc, char **argv){
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("CUDA first sample");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);
    //glDepthFunc(GL_ALWAYS);

    glViewport(0, 0, window_width, window_height);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

	//glEnable(GL_BLEND);
	//glBlendFunc(GL_SRC_ALPHA , GL_ONE);

    return true;
}

//////////////////////////////////////////////////////
////! Create VBO
void createVBOs(GLuint **local_vbo_list, struct hipGraphicsResource ***local_vbo_resources, int vbo_num, unsigned int size, unsigned int vbo_res_flags){
    *local_vbo_list = (GLuint *)malloc(sizeof(GLuint)*vbo_num);
    *local_vbo_resources = (struct hipGraphicsResource **)malloc( sizeof(struct hipGraphicsResource *)*vbo_num );


    glGenBuffers(vbo_num, *local_vbo_list );
    for(int i = 0; i < vbo_num; i++){
        glBindBuffer(GL_ARRAY_BUFFER, (*local_vbo_list)[i]);
        glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
        glBindBuffer(GL_ARRAY_BUFFER, 0);
        checkCudaErrors( hipGraphicsGLRegisterBuffer( &(*local_vbo_resources)[i], (*local_vbo_list)[i], vbo_res_flags ) );
    }

    return;
}

////! Delete VBO
void deleteVBOs(GLuint *vbo, struct hipGraphicsResource **vbo_res, int vbo_num){
    for(int i = 0; i < vbo_num; i++){
        hipGraphicsUnregisterResource(vbo_res[i]);
        glBindBuffer(1, vbo[i]);
        glDeleteBuffers(1, &vbo[i]);
        vbo[i] = 0;
    }
    return;
}

/////! Draw box line
int edge[12][2] = {
    {0,1},{1,2},{2,3},{3,0},
    {4,5},{5,6},{6,7},{7,4},
    {0,4},{1,5},{2,6},{3,7}
};
GLfloat frame_pos[8][3];
void draw_box_wire( GLfloat pos[8][3] ){

    glLineWidth(0.1f);
    glColor4f(1.0, 1.0, 1.0, 1.0f);
    glBegin(GL_LINES);
    for(int i=0;i<12;i++){
        glVertex3fv( pos[ edge[i][0] ] );
        glVertex3fv( pos[ edge[i][1] ] );
    }
    glEnd();
    glLineWidth(1.f);

}

//////////////////////////////////////////////////////
float g_fAnim = 0.f;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

float fov = 60.0f;
int mouse_buttons = 0;
int mouse_old_x, mouse_old_y;
float cen_dy = 0.f;

float *d_activity;
char *d_spike;
char h_spike[NUM_OF_NEURONS * PRE_LOAD * 2];
FILE *display_fp;
int display_isFirst = true;

char *d_type;
int3 *d_cmap;
int3 h_cmap[] = { {255,51,51}, {255,153,51},{255,255,51},{153,255,51},{51,255,51},{51,255,153},{51,255,255},{51,153,255},{51,51,255},{153,51,255},{255,51,255},{255,51,153},{160,160,160} };

int sub_t = 0;
pthread_t p_th[2];
hipStream_t stream[2];

const float view_port_radius = 10.f;
float phi = M_PI*3/5;
float theta = 0; //M_PI*3/5;
float middle[3] = {0.f,0.f,0.f};

void display2(){


    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT);
    glEnable(GL_DEPTH_TEST);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glViewport(0, 0, window_width, window_height);
    gluPerspective( fov, (GLfloat)window_width/(GLfloat) window_height, 0.1, 100.0);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    middle[0] = (frame_pos[0][0]+frame_pos[6][0])/2;
    middle[1] = (frame_pos[0][1]+frame_pos[6][1])/2;
    middle[2] = (frame_pos[0][2]+frame_pos[6][2])/2;

    gluLookAt(  middle[0]+view_port_radius*cosf(phi)*cosf(theta) , middle[1]+view_port_radius*sinf(theta), middle[2]+view_port_radius*sinf(phi)*cosf(theta),
                middle[0], middle[1] + cen_dy, middle[2],
                0.,1.,0.);


    glEnable(GL_ALPHA_TEST);
    //run CUDA
    hipGraphicsMapResources(2, vbo_resources, 0);
    float4 *pos, *col;
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer( (void **)&pos, &num_bytes, vbo_resources[0] );
    hipGraphicsResourceGetMappedPointer( (void **)&col, &num_bytes, vbo_resources[1] );

    if( display_isFirst ){
        hipMalloc( &d_activity, sizeof(float)*NUM_OF_NEURONS ) ;
        hipMalloc( &d_spike, sizeof(char)*NUM_OF_NEURONS*PRE_LOAD*2 );
        hipMalloc( &d_type, sizeof(char)*NUM_OF_NEURONS*PRE_LOAD );
        hipMalloc( &d_cmap, sizeof(int3)*13 );

        hipMemcpy( d_cmap, h_cmap, sizeof(int3)*13, hipMemcpyHostToDevice );

        float4 *h_pos = (float4 *)malloc(sizeof(float4)*NUM_OF_NEURONS);
        char *h_type = (char *)malloc(sizeof(char)*NUM_OF_NEURONS);
        fprintf(stderr, "h_pos: %p\n", h_pos);
        LoadPos( h_pos, h_type, frame_pos, "positions.dat", NUM_OF_NEURONS);

        hipMemcpy( pos, h_pos, sizeof(float4)*NUM_OF_NEURONS, hipMemcpyHostToDevice );
        hipMemcpy( d_type, h_type, sizeof(char)*NUM_OF_NEURONS, hipMemcpyHostToDevice );

        hipStreamCreateWithFlags(&stream[0], hipStreamNonBlocking);
        hipStreamCreateWithFlags(&stream[1], hipStreamNonBlocking);


        init_act_spike<<< (NUM_OF_NEURONS + 127)/128, 128>>>( d_activity, d_spike, NUM_OF_NEURONS );

        display_fp = fopen( "spike_data/spike.dat", "r");
        if(display_fp == NULL)exit(1);

        LoadSpike( &display_fp, h_spike, NUM_OF_NEURONS, PRE_LOAD);
        hipMemcpy( &d_spike, &h_spike, sizeof(char)*NUM_OF_NEURONS*PRE_LOAD, hipMemcpyHostToDevice );
        fprintf(stderr, "init_done\n");


        display_isFirst = false;
        //free(h_pos);
    }


    // load spike
    if( sub_t % PRE_LOAD == 0 && g_fAnim > 0){
        fprintf(stderr, "invoke sync\n");
        int i = !(sub_t/PRE_LOAD);
        hipStreamSynchronize(stream[i]);
        fprintf(stderr, "synchronized.\n");

        fprintf(stderr, "///////////////////////////// %lf %d %d %d\n", g_fAnim, PRE_LOAD , sub_t/PRE_LOAD, i);
        LoadSpike( &display_fp, &h_spike[i*NUM_OF_NEURONS*PRE_LOAD], NUM_OF_NEURONS, PRE_LOAD);
        hipMemcpyAsync( &d_spike[ i*NUM_OF_NEURONS*PRE_LOAD ], &h_spike[ i*NUM_OF_NEURONS*PRE_LOAD ], sizeof(char)*NUM_OF_NEURONS*PRE_LOAD, hipMemcpyHostToDevice, stream[i] );
    }

    update_activity<<< (NUM_OF_NEURONS+127)/128, 128 >>>(col, d_type, d_cmap, d_activity, d_spike, NUM_OF_NEURONS, sub_t);

    hipGraphicsUnmapResources(2, vbo_resources, 0);
    
    glBindBuffer(GL_ARRAY_BUFFER, vbo[1]);
    glColorPointer(4, GL_FLOAT, 0, 0);

    glBindBuffer(GL_ARRAY_BUFFER, vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);

    glDrawArrays(GL_POINTS, 0, NUM_OF_NEURONS );

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);

    draw_box_wire( frame_pos );


    glDisable(GL_ALPHA_TEST);
    glDisable(GL_DEPTH_TEST);
    glutSwapBuffers();
    g_fAnim += 0.01f;
    sub_t = (sub_t + 1 < PRE_LOAD*2)? sub_t + 1 : 0;
}

void display(){
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    gluLookAt(0.0,0.7,0.7, 0.,0.,0.,0.,1.,0.);

    //run CUDA
    hipGraphicsMapResources(2, vbo_resources, 0);
    float4 *pos, *col;
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer( (void **)&pos, &num_bytes, vbo_resources[0] );
    hipGraphicsResourceGetMappedPointer( (void **)&col, &num_bytes, vbo_resources[1] );


    dim3 block(8, 8, 1);
    dim3 grid(mesh_width/ block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block >>>( pos, col, mesh_width, mesh_height, g_fAnim);
    fprintf(stderr, "\r%lf", g_fAnim );

    hipGraphicsUnmapResources(2, vbo_resources, 0);
    
    glBindBuffer(GL_ARRAY_BUFFER, vbo[1]);
    glColorPointer(4, GL_FLOAT, 0, 0);

    glBindBuffer(GL_ARRAY_BUFFER, vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);


    glutSwapBuffers();
    g_fAnim += 0.01f;
}


void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void keyboard(unsigned char key, int , int ){
    switch(key){
        case 119: //w
            theta += M_PI/180*D_ANGLE;
            break;
        case 97: // a
            phi += M_PI/180*D_ANGLE;
            break;
        case 115: // s
            theta -= M_PI/180*D_ANGLE;
            break;
        case 100: // d
            phi -= M_PI/180*D_ANGLE;
            break;
        default:
            fprintf(stderr,"key: %d\n", key);
            break;
    }
}
void mouse(int button, int state, int x, int y)
{
    switch(button){
        case 3: // up
            fov -= 1.0f;
            break;
        case 4: // down
            fov += 1.0f;
            break;
    }
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
        cen_dy = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y){
    float dx, dy;
    dy = (float)(y - mouse_old_y);
    if( mouse_buttons & 1){
        cen_dy += dy * 0.01f;
    }else{
        cen_dy = 0;
    }
    mouse_old_x = x;
    mouse_old_y = y;
}

